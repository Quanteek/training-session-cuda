#include <fstream>
#include <iostream>
#include <stdexcept>
#include <ctime>
#include <hip/hip_runtime.h>
#include "kernel.cu"

int main(int argc, char ** argv)
{
  hipSetDevice(0);
  //Mise en place des donnees sur le host
  unsigned int nbr_data = 1024;
  unsigned int mem_size = sizeof(float)*nbr_data;
  float * h_idata = (float*)malloc(mem_size);
  for(int i = 0 ; i < nbr_data ; ++i)
    {
      h_idata[i] = 1;
    }

  //Allocation des espaces sur le device et copie du host sur le device
  float * d_idata;
  hipMalloc((void**)&d_idata, mem_size);
  hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice);

  float * d_odata;
  hipMalloc((void**)&d_odata, mem_size);

  //Mise en place du decoupage en fixant le nombre de threads
  unsigned int nbr_threads = 32;

  //unsigned int shared_mem_size = sizeof(float)*nbr_threads;

  //Calcul du nombre de blocs necessaires en fonction du nombre de threads et du nombre de donnees
  unsigned int nbr_blocks = (nbr_data+nbr_threads-1)/nbr_threads;

  //Allocation des grilles et threads
  dim3 grid(nbr_blocks, 1, 1);
  dim3 threads(nbr_threads, 1, 1);

  kernel<<<grid, threads>>>(d_idata, d_odata);

  hipMemcpy(h_idata, d_odata, mem_size, hipMemcpyDeviceToHost);
  
  std::ofstream file("./output.txt");
  if(!file.is_open())
    throw std::runtime_error("file error !"); 
  for(int i = 0 ; i < nbr_data ; ++i)
    {
      file << h_idata[i] << "\n";
    }
  file.close();
  
  hipFree(d_odata);
  hipFree(d_idata);
  free(h_idata);
  
  hipDeviceReset();
  return 0;
}
